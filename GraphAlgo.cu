#include "hip/hip_runtime.h"
/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-3
 * Description: Activation Game 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include "graph.hpp"
 
using namespace std;


ofstream outfile; // The handle for printing the output

/******************************Write your kerenels here ************************************/

//this kernel counts the number of nodes in the 0th level by finding change in apr value from zero to non zero
__global__ void countKernel(int* d_apr,int* dnodes_cur,int len){
    int id=blockIdx.x*1024+threadIdx.x*32+threadIdx.y;
    if(id<len && id!=0 && d_apr[id]!=0 && d_apr[id-1]==0)
        *dnodes_cur=id-1;
}

// this kernel is basically launched for number of nodes in a particular level
__global__ void activecountKernel(int* d_aid,int* d_apr,int* dnodes_cur,int* d_activeVertex,int* d_offset,int* d_csrList,int l,int len,int start){
    int id=blockIdx.x*1024+threadIdx.x*32+threadIdx.y;
    if(id<len)
    {
        //check if its one of the corner nodes of the level, and if not check either one of the neighbour is activated
        if(id==0 || id==len-1 || ( *(d_aid+start+id-1) >= *(d_apr+start+id-1) || *(d_aid+start+id+1) >= *(d_apr+start+id+1) ))
        {
            //now check if the node "id" itself is activated or not
            if( *(d_aid+start+id) >= *(d_apr+start+id) )
            {
                //if it is activated then increase the d_aid value of all nodes pointed by this node
                for(int i = 0; i < d_offset[start+id+1] - d_offset[start+id]; i++)
                {
                    //offset stores the value of node pointed by the activated node "id"
                    int offset=*(d_csrList+*(d_offset+id+start)+i);
                    //atomically increment value because different nodes can try to increment at the same time
                    atomicAdd(d_aid+offset,1);
                }
                //count the node "id" as activated nodes of this level
                atomicAdd(d_activeVertex+l,1);
            }
        }
        //now for all the nodes pointed by the nodes in level "l" store the value of maximum node to count length of next level
        for(int i=0;i<d_offset[start+id+1]-d_offset[start+id];i++)
        {
            //val stores the index of nodes pointed by node "id"
            int val=*(d_csrList+*(d_offset+id+start)+i);
            atomicMax(dnodes_cur,val);
        }
    }
}

/**************************************END*************************************************/



//Function to write result in output file
void printResult(int *arr, int V,  char* filename){
    outfile.open(filename);
    for(long int i = 0; i < V; i++){
        outfile<<arr[i]<<" ";   
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    int V ; // Number of vertices in the graph
    int E; // Number of edges in the graph
    int L; // number of levels in the graph

    //Reading input graph
    char *inputFilePath = argv[1];
    graph g(inputFilePath);

    //Parsing the graph to create csr list
    g.parseGraph();

    //Reading graph info 
    V = g.num_nodes();
    E = g.num_edges();
    L = g.get_level();


    //Variable for CSR format on host
    int *h_offset; // for csr offset
    int *h_csrList; // for csr
    int *h_apr; // active point requirement

    //reading csr
    h_offset = g.get_offset();
    h_csrList = g.get_csr();   
    h_apr = g.get_aprArray();
    
    // Variables for CSR on device
    int *d_offset;
    int *d_csrList;
    int *d_apr; //activation point requirement array
    int *d_aid; // acive in-degree array
    //Allocating memory on device 
    hipMalloc(&d_offset, (V+1)*sizeof(int));
    hipMalloc(&d_csrList, E*sizeof(int)); 
    hipMalloc(&d_apr, V*sizeof(int)); 
    hipMalloc(&d_aid, V*sizeof(int));

    //copy the csr offset, csrlist and apr array to device
    hipMemcpy(d_offset, h_offset, (V+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrList, h_csrList, E*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_apr, h_apr, V*sizeof(int), hipMemcpyHostToDevice);

    // variable for result, storing number of active vertices at each level, on host
    int *h_activeVertex;
    h_activeVertex = (int*)malloc(L*sizeof(int));
    // setting initially all to zero
    memset(h_activeVertex, 0, L*sizeof(int));

    // variable for result, storing number of active vertices at each level, on device
    int *d_activeVertex;
	hipMalloc(&d_activeVertex, L*sizeof(int));
    hipMemcpy(d_activeVertex, h_activeVertex, L*sizeof(int), hipMemcpyHostToDevice);


/***Important***/

    // Initialize d_aid array to zero for each vertex
    hipMemset(d_aid, 0, V*sizeof(int));
    // Make sure to use comments

/***END***/
double starttime = rtclock(); 

/*********************************CODE AREA*****************************************/
// len variable to launch kernell with maximum 10000 because a level has atmax 10000 nodes
int len=(V>10000)?10001:V;
//hnodes_cur stores the index of max node in current level
int *dnodes_cur;
//dnodes_cur for GPU memory
int *hnodes_cur;
hipMalloc(&dnodes_cur, sizeof(int));
hnodes_cur=(int*)malloc(sizeof(int));

dim3 gridDim(ceil(float(len)/1024),1,1);  
dim3 blockDim(32,32,1);	
//finds and stores the index of the last node of 0th level in dnodes_cur
countKernel<<<gridDim,blockDim>>>(d_apr,dnodes_cur,len);
hipMemcpy(hnodes_cur,dnodes_cur,sizeof(int),hipMemcpyDeviceToHost);

int start,l;
start=0;
l=0;
while(l<L){
    // len stores the number of nodes in level "l"
    len=*hnodes_cur-start+1;
    dim3 gridDim(ceil(float(len)/1024),1,1);  
    dim3 blockDim(32,32,1);
    activecountKernel<<<gridDim,blockDim>>>(d_aid,d_apr,dnodes_cur,d_activeVertex,d_offset,d_csrList,l,len,start);
    //update the value of start for next iteration before hnodes_cur gets updated
    start=*hnodes_cur+1;
    hipMemcpy(hnodes_cur,dnodes_cur,sizeof(int),hipMemcpyDeviceToHost);
    l++;
}
//stores back the value of d_activeVertex in h_activeVertex after all levels are done
hipMemcpy(h_activeVertex,d_activeVertex,L*sizeof(int),hipMemcpyDeviceToHost);

/********************************END OF CODE AREA**********************************/
double endtime = rtclock();  
printtime("GPU Kernel time: ", starttime, endtime);  

// --> Copy C from Device to Host
char outFIle[30] = "./output.txt" ;
printResult(h_activeVertex, L, outFIle);
if(argc>2)
{
    for(int i=0; i<L; i++)
    {
        printf("level = %d , active nodes = %d\n",i,h_activeVertex[i]);
    }
}

    return 0;
}
